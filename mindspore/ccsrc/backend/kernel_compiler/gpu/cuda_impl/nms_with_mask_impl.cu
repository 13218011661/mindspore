#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, softwareg
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "nms_with_mask_impl.cuh"
#include <limits>
#include <algorithm>

int NMSRoundUpPower2(int v) {
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;
  return v;
}

template <typename T>
__inline__ __device__ void Swap(T *lhs, T *rhs) {
  T tmp = lhs[0];
  lhs[0] = rhs[0];
  rhs[0] = tmp;
}

template <typename T>
__global__ void PopulateOutput(T *data_in, T *data_out, int *index_buff, const int num, int box_size_) {
  for (int box_num = blockIdx.x * blockDim.x + threadIdx.x; box_num < num; box_num += blockDim.x * gridDim.x) {
    int correct_index = index_buff[(num - 1) - box_num];  // flip the array around
    for (int x = 0; x < 5; x++) {
      data_out[(box_num * box_size_) + x] = data_in[(correct_index * box_size_) + x];
    }
  }
}

template <typename T>
__inline__ __device__ bool IOUDecision(T *output, int box_A_ix, int box_B_ix, int box_A_start, int box_B_start, T *area,
                                       float IOU_value) {
  T x_1 = max(output[box_A_start + 0], output[box_B_start + 0]);
  T y_1 = max(output[box_A_start + 1], output[box_B_start + 1]);
  T x_2 = min(output[box_A_start + 2], output[box_B_start + 2]);
  T y_2 = min(output[box_A_start + 3], output[box_B_start + 3]);
  T width = max(x_2 - x_1, T(0));  // in case of no overlap
  T height = max(y_2 - y_1, T(0));
  T combined_area = area[box_A_ix] + area[box_B_ix];
  // return decision to keep or remove box
  return !(((width * height) / (combined_area - (width * height))) > IOU_value);
}

template <typename T>
__global__ void Preprocess(const int num, int *sel_idx, T *area, T *output, int box_size_) {
  for (int box_num = blockIdx.x * blockDim.x + threadIdx.x; box_num < num; box_num += blockDim.x * gridDim.x) {
    sel_idx[box_num] = box_num;
    area[box_num] = (output[(box_num * box_size_) + 2] - output[(box_num * box_size_) + 0]) *
                    (output[(box_num * box_size_) + 3] - output[(box_num * box_size_) + 1]);
  }
}

template <typename T>
__global__ void NMSWithMaskKernel(const int num, const float IOU_value, T *output, T *area, bool *sel_boxes,
                                  int box_size_) {
  for (int box_num = blockIdx.x * blockDim.x + threadIdx.x; box_num < num; box_num += blockDim.x * gridDim.x) {
    // represents highest score box in that GPU block
    if (threadIdx.x == 0) {
      sel_boxes[box_num] = true;
      continue;
    }
    int box_start_index = box_num * box_size_;  // start index adjustment
    int block_max_box_num = ((blockIdx.x * blockDim.x) + 0);
    int block_max_box_start_index = block_max_box_num * box_size_;  // start index adjustment
    sel_boxes[box_num] =
      IOUDecision(output, box_num, block_max_box_num, block_max_box_start_index, box_start_index, area,
                  IOU_value);  // update mask
  }
}

template <typename T>
__global__ void FinalPass(const int num, const float IOU_value, T *output, T *area, bool *sel_boxes, int box_size_) {
  int box_i, box_j;                          // access all shared mem meta data with these
  int box_i_start_index, box_j_start_index;  // actual input data indexing
  for (int i = 0; i < num - 1; i++) {
    box_i = i;
    box_i_start_index = box_i * box_size_;  // adjust starting index
    if (sel_boxes[box_i]) {
      for (int j = i + 1; j < num; j++) {
        box_j = j;
        box_j_start_index = box_j * box_size_;
        if (sel_boxes[box_j]) {
          sel_boxes[box_j] = IOUDecision(output, box_i, box_j, box_i_start_index, box_j_start_index, area, IOU_value);
        }
      }
    }
  }
}

template <typename T>
__global__ void NMS_BitonicSortByKeyKernel(const int outer, const int inner, const int ceil_power2, T *input,
                                           T *data_buff, int *index_buff, int box_size_) {
  for (int i = threadIdx.x; i < ceil_power2; i += blockDim.x) {
    data_buff[i] = (i < inner) ? input[(i * box_size_) + 4] : std::numeric_limits<T>::max();
    index_buff[i] = i;
  }
  __syncthreads();

  for (size_t i = 2; i <= ceil_power2; i <<= 1) {
    for (size_t j = (i >> 1); j > 0; j >>= 1) {
      for (size_t tid = threadIdx.x; tid < ceil_power2; tid += blockDim.x) {
        size_t tid_comp = tid ^ j;
        if (tid_comp > tid) {
          if ((tid & i) == 0) {
            if (data_buff[tid] > data_buff[tid_comp]) {
              Swap(&data_buff[tid], &data_buff[tid_comp]);
              Swap(&index_buff[tid], &index_buff[tid_comp]);
            }
          } else {
            if (data_buff[tid] < data_buff[tid_comp]) {
              Swap(&data_buff[tid], &data_buff[tid_comp]);
              Swap(&index_buff[tid], &index_buff[tid_comp]);
            }
          }
        }
      }
      __syncthreads();
    }
  }
}

template <typename T>
void CalPreprocess(const int num, int *sel_idx, T *area, T *input, T *output, int *index_buff, int box_size_,
                   hipStream_t hip_stream) {
  PopulateOutput<<<GET_BLOCKS(num), GET_THREADS, 0, hip_stream>>>(input, output, index_buff, num, box_size_);
  Preprocess<<<GET_BLOCKS(num), GET_THREADS, 0, hip_stream>>>(num, sel_idx, area, output, box_size_);
}

template <typename T>
void CalSortInit(const int &num, T *data_in, T *data_out, int *index_buff, T *data_buff, int box_size_,
                 hipStream_t stream) {
  int ceil_p_2 = NMSRoundUpPower2(num);
  int thread = std::min(ceil_p_2, GET_THREADS);
  NMS_BitonicSortByKeyKernel<<<1, thread, 0, stream>>>(1, num, ceil_p_2, data_in, data_buff, index_buff, box_size_);
}

template <typename T>
void CalNMSWithMask(const int num, const float IOU_value, T *output, T *area, bool *sel_boxes, int box_size_,
                    hipStream_t hip_stream) {
  NMSWithMaskKernel<<<GET_BLOCKS(num), GET_THREADS, 0, hip_stream>>>(num, IOU_value, output, area, sel_boxes,
                                                                      box_size_);
}

template <typename T>
void CalFinalPass(const int num, const float IOU_value, T *output, T *area, bool *sel_boxes, int box_size_,
                  hipStream_t hip_stream) {
  FinalPass<<<1, 1, 0, hip_stream>>>(num, IOU_value, output, area, sel_boxes, box_size_);
}

template void CalPreprocess<float>(const int num, int *sel_idx, float *area, float *input, float *output,
                                   int *index_buff, int box_size_, hipStream_t hip_stream);

template void CalSortInit<float>(const int &inner, float *data_in, float *data_out, int *index_buff, float *data_buff,
                                 int box_size_, hipStream_t stream);

template void CalNMSWithMask<float>(const int num, const float IOU_value, float *output, float *area, bool *sel_boxes,
                                    int box_size_, hipStream_t hip_stream);

template void CalFinalPass<float>(const int num, const float IOU_value, float *output, float *area, bool *sel_boxes,
                                  int box_size_, hipStream_t hip_stream);
