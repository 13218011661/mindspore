#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, softwareg
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "nms_with_mask_impl.cuh"
#include <limits>
#include <algorithm>

int NMSRoundUpPower2(int v) {
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;
  return v;
}

template <typename T>
__inline__ __device__ void Swap(T *lhs, T *rhs) {
  T tmp = lhs[0];
  lhs[0] = rhs[0];
  rhs[0] = tmp;
}

// Initialize per row mask array to all true
__global__ void MaskInit(int numSq, bool *row_mask) {
  for (int mat_pos = blockIdx.x * blockDim.x + threadIdx.x; mat_pos < numSq; mat_pos += blockDim.x * gridDim.x) {
    row_mask[mat_pos] = true;
  }
}

// copy data from input to output array sorted by indices returned from bitonic sort
// flips boxes if asked to,  default - false -> if (x1/y1 > x2/y2)
template <typename T>
__global__ void PopulateOutput(T *data_in, T *data_out, int *index_buff, const int num, int box_size_,
                               bool flip_mode = false) {
  for (int box_num = blockIdx.x * blockDim.x + threadIdx.x; box_num < num; box_num += blockDim.x * gridDim.x) {
    int correct_index = index_buff[(num - 1) - box_num];  // flip the array around
    int correct_arr_start = correct_index * box_size_;
    int current_arr_start = box_num * box_size_;
    if (flip_mode) {  // flip boxes
      // check x
      if (data_in[correct_arr_start + 0] > data_in[correct_arr_start + 2]) {
        data_out[current_arr_start + 0] = data_in[correct_arr_start + 2];
        data_out[current_arr_start + 2] = data_in[correct_arr_start + 0];
      } else {
        data_out[current_arr_start + 0] = data_in[correct_arr_start + 0];
        data_out[current_arr_start + 2] = data_in[correct_arr_start + 2];
      }
      // check y
      if (data_in[correct_arr_start + 1] > data_in[correct_arr_start + 3]) {
        data_out[current_arr_start + 1] = data_in[correct_arr_start + 3];
        data_out[current_arr_start + 3] = data_in[correct_arr_start + 1];
      } else {
        data_out[current_arr_start + 1] = data_in[correct_arr_start + 1];
        data_out[current_arr_start + 3] = data_in[correct_arr_start + 3];
      }
      data_out[current_arr_start + 4] = data_in[correct_arr_start + 4];
    } else {  // default behaviour, don't flip
      for (int x = 0; x < 5; x++) {
        data_out[current_arr_start + x] = data_in[correct_arr_start + x];
      }
    }
  }
}

template <typename T>
__inline__ __device__ bool IOUDecision(T *output, int box_A_ix, int box_B_ix, int box_A_start, int box_B_start, T *area,
                                       float IOU_value) {
  T x_1 = max(output[box_A_start + 0], output[box_B_start + 0]);
  T y_1 = max(output[box_A_start + 1], output[box_B_start + 1]);
  T x_2 = min(output[box_A_start + 2], output[box_B_start + 2]);
  T y_2 = min(output[box_A_start + 3], output[box_B_start + 3]);
  T width = max(x_2 - x_1, T(0));  // in case of no overlap
  T height = max(y_2 - y_1, T(0));
  T combined_area = area[box_A_ix] + area[box_B_ix];
  // return decision to keep or remove box
  return !(((width * height) / (combined_area - (width * height))) >= IOU_value);
}

// calculate areas for boxes -> sorted by output boxes
// populated return mask (init to all true) and return index array
template <typename T>
__global__ void Preprocess(const int num, int *sel_idx, bool *sel_boxes, T *area, T *output, int box_size_) {
  for (int box_num = blockIdx.x * blockDim.x + threadIdx.x; box_num < num; box_num += blockDim.x * gridDim.x) {
    sel_idx[box_num] = box_num;
    sel_boxes[box_num] = true;
    area[box_num] = (output[(box_num * box_size_) + 2] - output[(box_num * box_size_) + 0]) *
                    (output[(box_num * box_size_) + 3] - output[(box_num * box_size_) + 1]);
  }
}

// Run parallel NMS pass
// Every box updates it's own mask in row_mask in sep threads
template <typename T>
__global__ void NMSPass(const int num, const float IOU_value, T *output, T *area, bool *sel_boxes, int box_size_,
                        bool *row_mask) {
  int box_i_start_index, box_j_start_index;  // actual input data indexing
  int mask_offset = 0;
  for (int box_i = blockIdx.x * blockDim.x + threadIdx.x; box_i < num - 1; box_i += blockDim.x * gridDim.x) {
    mask_offset = box_i * num;
    box_i_start_index = box_i * box_size_;  // adjust starting index
    for (int box_j = box_i + 1; box_j < num; box_j++) {
      box_j_start_index = box_j * box_size_;
      row_mask[mask_offset + box_j] =
        IOUDecision(output, box_i, box_j, box_i_start_index, box_j_start_index, area, IOU_value);
    }
  }
}

// Reduce pass runs on 1 block to allow thread sync
__global__ void ReducePass(const int num, bool *sel_boxes, bool *row_mask) {
  // loop over every box in order of high to low confidence score
  for (int i = 0; i < num - 1; ++i) {
    if (!sel_boxes[i]) {
      continue;
    }
    // every thread handles a different set of boxes (per all boxes in order)
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < num; j += blockDim.x * gridDim.x) {
      sel_boxes[j] = sel_boxes[j] && row_mask[i * num + j];
    }
    __syncthreads();  // sync all threads before moving all active threads to next iteration
  }
}

// Sorting function based on BitonicSort from TopK kernel
template <typename T>
__global__ void NMS_BitonicSortByKeyKernel(const int outer, const int inner, const int ceil_power2, T *input,
                                           T *data_buff, int *index_buff, int box_size_) {
  for (int i = threadIdx.x; i < ceil_power2; i += blockDim.x) {
    data_buff[i] = (i < inner) ? input[(i * box_size_) + 4] : std::numeric_limits<T>::max();
    index_buff[i] = i;
  }
  __syncthreads();

  for (size_t i = 2; i <= ceil_power2; i <<= 1) {
    for (size_t j = (i >> 1); j > 0; j >>= 1) {
      for (size_t tid = threadIdx.x; tid < ceil_power2; tid += blockDim.x) {
        size_t tid_comp = tid ^ j;
        if (tid_comp > tid) {
          if ((tid & i) == 0) {
            if (data_buff[tid] > data_buff[tid_comp]) {
              Swap(&data_buff[tid], &data_buff[tid_comp]);
              Swap(&index_buff[tid], &index_buff[tid_comp]);
            }
          } else {
            if (data_buff[tid] < data_buff[tid_comp]) {
              Swap(&data_buff[tid], &data_buff[tid_comp]);
              Swap(&index_buff[tid], &index_buff[tid_comp]);
            }
          }
        }
      }
      __syncthreads();
    }
  }
}

template <typename T>
void CalPreprocess(const int num, int *sel_idx, bool *sel_boxes, T *area, T *input, T *output, int *index_buff,
                   int box_size_, bool *row_mask, hipStream_t hip_stream) {
  int total_val = num * num;
  MaskInit<<<GET_BLOCKS(total_val), GET_THREADS, 0, hip_stream>>>(total_val, row_mask);
  // default for flipping boxes -> false (provision available to flip if API updated)
  PopulateOutput<<<GET_BLOCKS(num), GET_THREADS, 0, hip_stream>>>(input, output, index_buff, num, box_size_, false);
  Preprocess<<<GET_BLOCKS(num), GET_THREADS, 0, hip_stream>>>(num, sel_idx, sel_boxes, area, output, box_size_);
}

template <typename T>
void CalSort(const int &num, T *data_in, T *data_out, int *index_buff, T *data_buff, int box_size_,
             hipStream_t stream) {
  int ceil_p_2 = NMSRoundUpPower2(num);
  int thread = std::min(ceil_p_2, GET_THREADS);
  NMS_BitonicSortByKeyKernel<<<1, thread, 0, stream>>>(1, num, ceil_p_2, data_in, data_buff, index_buff, box_size_);
}

template <typename T>
void CalNMS(const int num, const float IOU_value, T *output, T *area, bool *sel_boxes, int box_size_, bool *row_mask,
            hipStream_t hip_stream) {
  NMSPass<<<GET_BLOCKS(num), GET_THREADS, 0, hip_stream>>>(num, IOU_value, output, area, sel_boxes, box_size_,
                                                            row_mask);
  ReducePass<<<1, GET_THREADS, 0, hip_stream>>>(num, sel_boxes, row_mask);
}

template void CalSort<float>(const int &inner, float *data_in, float *data_out, int *index_buff, float *data_buff,
                             int box_size_, hipStream_t stream);

template void CalPreprocess<float>(const int num, int *sel_idx, bool *sel_boxes, float *area, float *input,
                                   float *output, int *index_buff, int box_size_, bool *row_mask,
                                   hipStream_t hip_stream);

template void CalNMS<float>(const int num, const float IOU_value, float *output, float *area, bool *sel_boxes,
                            int box_size_, bool *row_mask, hipStream_t hip_stream);
