#include "hip/hip_runtime.h"
/**
 * Copyright 2019 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "kernel/gpu/cuda_impl/concatv2_impl.cuh"
template <typename T>
__global__ void ConcatV2(const size_t size, const int w1, const int w2, const T* input_1, const T* input_2, T* output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    int n = pos / (w1 + w2);
    int m = pos % (w1 + w2);
    output[pos] = m >= w1 ? input_2[n * w2 + m - w1] : input_1[n * w1 + m];
  }
  return;
}

template <typename T>
void CalConcatV2(const size_t size, const int w1, const int w2, const T* input_1, const T* input_2, T* output,
                 hipStream_t hip_stream) {
  ConcatV2<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, w1, w2, input_1, input_2, output);
  return;
}

template void CalConcatV2(const size_t size, const int w1, const int w2, const float* input_1, const float* input_2,
                          float* output, hipStream_t hip_stream);
template void CalConcatV2(const size_t size, const int w1, const int w2, const int* input_1, const int* input_2,
                          int* output, hipStream_t hip_stream);
template void CalConcatV2(const size_t size, const int w1, const int w2, const half* input_1, const half* input_2,
                          half* output, hipStream_t hip_stream);

